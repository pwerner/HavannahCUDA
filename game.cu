/*	Havannah CUDA � This program provides a CUDA based playout strategy for Monte-Carlo-Evaluation
 *	Copyright (C) 2012  Peter Werner
 *	This program is free software: you can redistribute it and/or modify
 *	it under the terms of the GNU General Public License as published by
 *	the Free Software Foundation, either version 3 of the License, or
 *	(at your option) any later version.
 *
 *	This program is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU General Public License for more details.
 *
 *	You should have received a copy of the GNU General Public License
 *	along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *	Contact: Peter@wernerbrothers.de
 */

#include "game.h"

void win(game * g) {
	g->win = g->win+1;
}

void loss(game * g) {
	g->loss = g->loss+1;
}

void draw(game * g) {
	g->draw = g->draw+1;
}

void init_game(game* g, idx move) {
	g->move = move;
	g->win = 0;
	g->loss = 0;
	g->draw = 0;
}